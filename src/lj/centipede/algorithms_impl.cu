#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// Copyright (c) 2017 by Lukasz Janyst <lukasz@jany.st>
//------------------------------------------------------------------------------
// This file is part of the Centipede software suite.
//
// Centipede is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Centipede is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with Centipede.  If not, see <http://www.gnu.org/licenses/>.
//------------------------------------------------------------------------------

#include "algorithms_impl.hh"

namespace {

//------------------------------------------------------------------------------
// Transformation kernel
//------------------------------------------------------------------------------
__global__ void transform(void *output, const void *input,
                          uint32_t out_elem_size, uint32_t in_elem_size,
                          uint32_t size, lj::centipede::UnaryFunc op) {
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  const char *inputI  = (const char *)input;
  char       *outputI = (char *)output;
  if(idx >= size)
    return;

  inputI  += idx*in_elem_size;
  outputI += idx*out_elem_size;
  (*op)(outputI, inputI);
}
}

namespace lj { namespace centipede {

//------------------------------------------------------------------------------
// Transformation
//------------------------------------------------------------------------------
template<>
void TransformImpl<DeviceType::CUDA, AlgorithmType::Parallel>(
  void *output, uint32_t out_elem_size,
  const void *input, uint32_t in_elem_size,
  uint32_t size, UnaryFunc op) {

  void *d_input;
  void *d_output;

  uint32_t input_size  = size*in_elem_size;
  uint32_t output_size = size*out_elem_size;

  auto status = hipMalloc((void**) &d_input,  input_size);
  if(status != hipSuccess)
    throw std::runtime_error("Unable to allocate GPU memory for input data");

  status = hipMalloc((void**) &d_output, output_size);
  if(status != hipSuccess) {
    hipFree(d_input);
    throw std::runtime_error("Unable to allocate GPU memory for output data");
  }

  hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);

  uint32_t blocks = size/1024 + 1;
  transform<<<blocks, 1024>>>(d_output, d_input, out_elem_size, in_elem_size,
                              size, op);

  hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}

} }
